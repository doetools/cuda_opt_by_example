#include "data_structure.h"

// template <class T>
FLOAT measure_performance(std::function<void(hipStream_t)> bound_function,
                          hipStream_t stream, int num_repeats = 100,
                          int num_warmups = 100)
{
    hipEvent_t start, stop;
    FLOAT time;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    for (int i{0}; i < num_warmups; ++i)
    {
        bound_function(stream);
    }

    hipStreamSynchronize(stream);

    hipEventRecord(start, stream);
    for (int i{0}; i < num_repeats; ++i)
    {
        bound_function(stream);
    }
    hipEventRecord(stop, stream);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    FLOAT const latency{time / num_repeats};

    return latency;
}