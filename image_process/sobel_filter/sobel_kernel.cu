#include "hip/hip_runtime.h"
#include "../../cuda/data_structure.h"

template <typename T> __host__ __device__ T convolute(T *a, T *b, const int N) {
  T total{0};

  for (int i = 0; i < N; i++) {
    total += a[i] * b[N - 1 - i];
  }

  return T(total);
}

// kernels
__device__ const FLOAT KERNEL_X[9] = {1.0,  0.0, -1.0, 2.0, 0.0,
                                      -2.0, 1.0, 0.0,  -1.0};
__device__ const FLOAT KERNEL_Y[9] = {-1.0, -2.0, -1.0, 0,  0.0,
                                      0.0,  1.0,  2.0,  1.0};

template <class T, const int WARP_SIZE = 32>
__global__ void sobel_convolute_naive(const T *img, T *new_img, const size_t M,
                                      const size_t N) {

  size_t gj{blockIdx.x * blockDim.x + threadIdx.x};
  size_t gi{blockIdx.y * blockDim.y + threadIdx.y};

  size_t j = threadIdx.x;
  size_t i = threadIdx.y;

  int KERNEL_SIZE{3};
  int KERNEL_LEN{9};

  // a 32+2 by 32+2 tile
  __shared__ T img_tile[WARP_SIZE + 2][WARP_SIZE + 2];

  // read data to shared memory
  img_tile[i + 1][j + 1] = img[gi * N + gj];

  // read left
  if (blockIdx.x != 0) {
    if (j == 0)
      img_tile[i + 1][j] = img[gi * N + gj - 1];
  }

  // read right
  if (blockIdx.x != gridDim.x - 1)
    if (j == WARP_SIZE - 1)
      img_tile[i + 1][j + 2] = img[gi * N + gj + 1];

  // read bottom
  if (blockIdx.y != 0) {
    if (i == 0)
      img_tile[i][j + 1] = img[(gi - 1) * N + gj];
  }

  // read top
  if (blockIdx.y != gridDim.y - 1) {
    if (i == WARP_SIZE - 1)
      img_tile[i + 2][j + 1] = img[(gi + 1) * N + gj];
  }

  // four corners
  if (i == 0 && j == 0) {
    if ((gi - 1) > 0 && (gj - 1) > 0) {
      img_tile[0][0] = img[(gi - 1) * N + gj - 1];
    }
  }
  if (i == 0 && j == WARP_SIZE - 1) {
    if ((gi - 1) > 0 && (gj + 1) < N) {

      img_tile[0][WARP_SIZE + 1] = img[(gi - 1) * N + gj + 1];
    }
  }
  if (i == WARP_SIZE - 1 && j == 0) {
    if ((gi + 1) < M && (gj - 1) > 0) {

      img_tile[WARP_SIZE + 1][0] = img[(gi + 1) * N + gj - 1];
    }
  }
  if (i == WARP_SIZE - 1 && j == WARP_SIZE - 1) {
    if ((gi + 1) < M && (gj + 1) < N) {
      img_tile[WARP_SIZE + 1][WARP_SIZE + 1] = img[(gi + 1) * N + gj + 1];
    }
  }

  __syncthreads();

  if (gi == 0 || gi >= M - 1 || gj == 0 || gj >= N - 1)
    return;

  // calculate sobel
  int index;
  int increment_i = 0, increment_j = 0;

  FLOAT total_x = 0;
  for (index = 0; index < KERNEL_LEN; index++) {
    increment_i = index / KERNEL_SIZE;
    increment_j = index % KERNEL_SIZE;
    total_x += img_tile[i + increment_i][j + increment_j] *
               KERNEL_X[KERNEL_LEN - 1 - index];
  }

  FLOAT total_y = 0;
  for (index = 0; index < KERNEL_LEN; index++) {
    increment_i = index / KERNEL_SIZE;
    increment_j = index % KERNEL_SIZE;
    total_y += img_tile[i + increment_i][j + increment_j] *
               KERNEL_Y[KERNEL_LEN - 1 - index];
  }

  // save result to new_img of M-2 by N-2
  new_img[gi * N + gj] = sqrtf(total_x * total_x + total_y * total_y);

  if (gi == 1 && gj == 1) {
    printf("final: %f\n", sqrtf(total_x * total_x + total_y * total_y));
  }
}